#include "hip/hip_runtime.h"
﻿__kernel void <DTYPE_NAME>_add(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] + y[i];
}

__kernel void <DTYPE_NAME>_subtract(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] - y[i];
}

__kernel void <DTYPE_NAME>_multiply(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] * y[i];
}

__kernel void <DTYPE_NAME>_divide(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] / y[i];
}

__kernel void <DTYPE_NAME>_remainder(__global <DTYPE_NAME> *x1, __global <DTYPE_NAME> *x2, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = remainder(x1[i], x2[i]);
}

__kernel void <DTYPE_NAME>_gt(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    if(x[i] > y[i]){
        r[i] = 1;
    }
    else{
        r[i] = 0;
    }
}

__kernel void <DTYPE_NAME>_ge(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    if(x[i] >= y[i]){
        r[i] = 1;
    }
    else{
        r[i] = 0;
    }
}

__kernel void <DTYPE_NAME>_lt(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    if(x[i] < y[i]){
        r[i] = 1;
    }
    else{
        r[i] = 0;
    }
}

__kernel void <DTYPE_NAME>_le(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    if(x[i] <= y[i]){
        r[i] = 1;
    }
    else{
        r[i] = 0;
    }
}

__kernel void <DTYPE_NAME>_eq(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    if(x[i] == y[i]){
        r[i] = 1;
    }
    else{
        r[i] = 0;
    }
}

__kernel void <DTYPE_NAME>_noteq(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    if(x[i] != y[i]){
        r[i] = 1;
    }
    else{
        r[i] = 0;
    }
}

__kernel void <DTYPE_NAME>_bitwise_and(__global int *x, __global int *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] & y[i];
}

__kernel void <DTYPE_NAME>_bitwise_or(__global int *x, __global int *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] | y[i];
}

__kernel void <DTYPE_NAME>_bitwise_xor(__global int *x, __global int *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] ^ y[i];
}

__kernel void <DTYPE_NAME>_bitwise_not(__global int* x, __global <DTYPE_NAME>* r)
{
    int i = get_global_id(0);
    r[i] = ~x[i];
}

__kernel void <DTYPE_NAME>_logical_and(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] && y[i];
}

__kernel void <DTYPE_NAME>_logical_or(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] || y[i];
}

/*
__kernel void <DTYPE_NAME>_logical_xor(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] ^ y[i];
}
*/

__kernel void <DTYPE_NAME>_left_shift(__global int* x, __global int *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] << y[i];
}

__kernel void <DTYPE_NAME>_right_shift(__global int *x, __global int *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] >> y[i];
}

__kernel void <DTYPE_NAME>_floor_divide(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = floor(x[i] / y[i]);
}

__kernel void <DTYPE_NAME>_logaddexp(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *y, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = log(exp(x[i]) + exp(y[i]));
}

__kernel void <DTYPE_NAME>_abs(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = fabs(x[i]);
}

__kernel void <DTYPE_NAME>_acos(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = acos(x[i]);
}

__kernel void <DTYPE_NAME>_acosh(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = acosh(x[i]);
}

__kernel void <DTYPE_NAME>_asin(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = asin(x[i]);
}

__kernel void <DTYPE_NAME>_asinh(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = asinh(x[i]);
}

__kernel void <DTYPE_NAME>_atan(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = atan(x[i]);
}

__kernel void <DTYPE_NAME>_atan2(__global <DTYPE_NAME> *y, __global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = atan2(y[i], x[i]);
}

__kernel void <DTYPE_NAME>_atanh(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = atanh(x[i]);
}

__kernel void <DTYPE_NAME>_ceil(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = asinh(x[i]);
}

__kernel void <DTYPE_NAME>_cos(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = cos(x[i]);
}

__kernel void <DTYPE_NAME>_cosh(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = cosh(x[i]);
}

__kernel void <DTYPE_NAME>_exp(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = exp(x[i]);
}

__kernel void <DTYPE_NAME>_expm1(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = expm1(x[i]);
}

__kernel void <DTYPE_NAME>_floor(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = floor(x[i]);
}

__kernel void <DTYPE_NAME>_isfinite(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = isfinite(x[i]);
}

__kernel void <DTYPE_NAME>_isinf(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = isinf(x[i]);
}

__kernel void <DTYPE_NAME>_isnan(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = isnan(x[i]);
}

__kernel void <DTYPE_NAME>_log(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = log(x[i]);
}

__kernel void <DTYPE_NAME>_log1p(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = log1p(x[i]);
}

__kernel void <DTYPE_NAME>_log2(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = log2(x[i]);
}

__kernel void <DTYPE_NAME>_log10(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = log10(x[i]);
}

__kernel void <DTYPE_NAME>_negative(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = x[i] * -1;
}

__kernel void <DTYPE_NAME>_positive(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    if(x[i] < 0){
        r[i] = x[i] * -1;
    }
    else{
        r[i] = x[i];
    }
}

__kernel void <DTYPE_NAME>_power(__global <DTYPE_NAME> *x1, __global <DTYPE_NAME> *x2, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = pow(x1[i], x2[i]);
}

__kernel void <DTYPE_NAME>_round(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = round(x[i]);
}

__kernel void <DTYPE_NAME>_sign(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = sign(x[i]);
}

__kernel void <DTYPE_NAME>_sinh(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = sinh(x[i]);
}

__kernel void <DTYPE_NAME>_sin(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = sin(x[i]);
}

__kernel void <DTYPE_NAME>_square(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = pown(x[i], 2);
}

__kernel void <DTYPE_NAME>_sqrt(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = sqrt(x[i]);
}

__kernel void <DTYPE_NAME>_tan(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = tan(x[i]);
}

__kernel void <DTYPE_NAME>_tanh(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = tan(x[i]);
}

__kernel void <DTYPE_NAME>_trunc(__global <DTYPE_NAME> *x, __global <DTYPE_NAME> *r)
{
    int i = get_global_id(0);
    r[i] = trunc(x[i]);
}