﻿__kernel void <DTYPE_NAME>_matmul(const long M, const long N, const long K,
                      const __global <DTYPE_NAME>* A,
                      const __global <DTYPE_NAME>* B,
                      __global <DTYPE_NAME>* C) {
    
    // Thread identifiers
    const int globalRow = get_global_id(0); // Row ID of C (0..M)
    const int globalCol = get_global_id(1); // Col ID of C (0..N)
    
    // Compute a single element (loop over K)
    <DTYPE_NAME> acc = 0;
    for (int k=0; k<K; k++) {
        acc += A[k*M + globalRow] * B[globalCol*K + k];
    }
 
    // Store the result
    C[globalCol*M + globalRow] = acc;
}