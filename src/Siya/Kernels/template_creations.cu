﻿__kernel void <DTYPE_NAME>_full(double fill_value, __global <DTYPE_NAME>* r)
{
    int i = get_global_id(0);
    r[i] = fill_value;
}